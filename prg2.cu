#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "gpu_kernel.cuh"
#include "cpu_math.h"
#include "file_handler.h"
#include "config.h"

void fill_matrix(float* matrix, int N);
void fill_vector(float* vector, int N);
void print_matrix_result(float* matrix, int N);
void print_vector_result(float* vector, int N);
void cpu_mode();
void gpu_mode();

int main(int argc, char* argv[])
{
    // Check if the program is run with the correct number of arguments
    if (argc != 2)
    {
        printf("Usage: %s <GPU_ENABLE>\n", argv[0]);
        return -1;
    }

    // Parse GPU_ENABLE from command line argument
    int GPU_ENABLE = atoi(argv[1]);

    // Check if GPU_ENABLE is valid
    if (GPU_ENABLE != 0 && GPU_ENABLE != 1)
    {
        printf("Error: GPU_ENABLE must be 0 or 1\n");
        return -1;
    }

    // Check CUBLAS_ENABLE from config.h
    if (CUBLAS_ENABLE != 0 && CUBLAS_ENABLE != 1)
    {
        printf("Error: CUBLAS_ENABLE must be 0 or 1\n");
        return -1;
    }

    // CPU mode memory allocation
    if (GPU_ENABLE == 0)
    {
        cpu_mode();
    }

    // GPU mode memory allocation
    if (GPU_ENABLE == 1)
    {
        gpu_mode();
    }

    printf("Done\n");
    return 0;

    return 0;
}

void fill_matrix(float* matrix, int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            matrix[i * N + j] = 0.0f;
        }
    }
}

void fill_vector(float* vector, int N)
{
    for (int i = 0; i < N; i++)
    {
        vector[i] = 0.0f;
    }
}

void print_matrix_result(float* matrix, int N)
{
    if (N > 32)
    {
        printf("Result is too large to print.\n");
        return;
    }

    printf("Result: ");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%f ", matrix[i * N + j]);
        }
        printf("\n");
    }

    printf("\n");
}

void print_vector_result(float* vector, int N)
{
    if (N > 32)
    {
        printf("Result is too large to print.\n");
        return;
    }

    printf("Result: ");
    for (int i = 0; i < N; i++)
    {
        printf("%f ", vector[i]);
    }

    printf("\n");
}

void cpu_mode()
{
    printf("CPU mode\n");
    // Variables
    int matrix_size = SIZE_N * SIZE_N * sizeof(float);
    int vector_size = SIZE_N * sizeof(float);
    float *A, *B, *MC, *MD, *E, *Mtmp;

    // Malloc
    A = (float*)malloc(vector_size);
    if (A == NULL)
    {
        printf("Error allocating memory for A\n");
        return;
    }
    B = (float*)malloc(vector_size);
    if (B == NULL)
    {
        printf("Error allocating memory for B\n");
        free(A);
        return;
    }
    MC = (float*)malloc(matrix_size);
    if (MC == NULL)
    {
        printf("Error allocating memory for MC\n");
        free(A);
        free(B);
        return;
    }
    MD = (float*)malloc(matrix_size);
    if (MD == NULL)
    {
        printf("Error allocating memory for MD\n");
        free(A);
        free(B);
        free(MC);
        return;
    }
    E = (float*)malloc(vector_size);
    if (E == NULL)
    {
        printf("Error allocating memory for E\n");
        free(A);
        free(B);
        free(MC);
        free(MD);
        return;
    }
    Mtmp = (float*)malloc(matrix_size);
    if (Mtmp == NULL)
    {
        printf("Error allocating memory for Mtmp\n");
        free(A);
        free(B);
        free(MC);
        free(MD);
        free(E);
        return;
    }


    // Fill A, Mtmp
    fill_vector(A, SIZE_N);
    fill_matrix(Mtmp, SIZE_N);

    // Read data from files
    read_vector_float("data\\256_B_f.txt", B, SIZE_N);
    read_matrix_float("data\\256_MC_f.txt", MC, SIZE_N, SIZE_N);
    read_matrix_float("data\\256_MD_f.txt", MD, SIZE_N, SIZE_N);
    read_vector_float("data\\256_E_f.txt", E, SIZE_N);

    // Timer + Start
    struct timespec start, end;
    timespec_get(&start, TIME_UTC);

    // A = B + MC * MD * E
    f_matrix_multiply_matrix_acc(MC, MD, Mtmp, SIZE_N);
    f_matrix_multiply_vector_acc(Mtmp, E, A, SIZE_N);
    f_vector_add_vector(A, B, SIZE_N);

    // End timer
    timespec_get(&end, TIME_UTC);

    // Show elapsed time
    double elapsed_time = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1e9;
    printf("Elapsed time: %.5f milliseconds\n", elapsed_time * 1000);

    // Print and save result
    print_vector_result(A, SIZE_N);
    write_vector_float("result\\result_cpu_prg2.txt", A, SIZE_N);

    // Free malloc
    free(A);
    free(B);
    free(MC);
    free(MD);
    free(E);
    free(Mtmp);
    A = NULL;
    B = NULL;
    MC = NULL;
    MD = NULL;
    E = NULL;
    Mtmp = NULL;

    return;
}

void gpu_mode()
{
    if (THREADS_PER_BLOCK > 1024)
    {
        printf("Error: THREADS_PER_BLOCK exceeds 1024\n");
        return;
    }
    if (GRID_Y > 65535 || BLOCKS_PER_GRID > 2147483647) // "2 ^ 31 - 1" or "(1 << 31) - 1"
    {
        printf("Error: BLOCK_Y exceeds 65535 or BLOCKS_PER_GRID exceeds 2 ^ 31 - 1\n");
        return;
    }

    printf("GPU mode\n");
    // Variables
    int matrix_size = SIZE_N * SIZE_N * sizeof(float);
    int vector_size = SIZE_N * sizeof(float);
    float *A, *B, *MC, *MD, *E, *Mtmp;
    hipError_t err;

    // CUDA events for performance measurement
    hipEvent_t start, stop;
    err = hipEventCreate(&start);
    if (err != hipSuccess)
    {
        printf("Error creating event for timing.");
        return;
    }
    err = hipEventCreate(&stop);
    if (err != hipSuccess)
    {
        printf("Error creating event for timing.");
        return;
    }

    hipStream_t stream;
    err = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    if (err != hipSuccess)
    {
        printf("Error creating stream.");
        return;
    }

    // Allocate memory (Host)
    err = hipHostMalloc((void**)&A, vector_size);
    if (err != hipSuccess)
    {
        printf("Error allocating memory for A\n");
        return;
    }
    err = hipHostMalloc((void**)&B, vector_size);
    if (err != hipSuccess)
    {
        printf("Error allocating memory for B\n");
        hipHostFree(A);
        return;
    }
    err = hipHostMalloc((void**)&MC, matrix_size);
    if (err != hipSuccess)
    {
        printf("Error allocating memory for MC\n");
        hipHostFree(A);
        hipHostFree(B);
        return;
    }
    err = hipHostMalloc((void**)&MD, matrix_size);
    if (err != hipSuccess)
    {
        printf("Error allocating memory for MD\n");
        hipHostFree(A);
        hipHostFree(B);
        hipHostFree(MC);
        return;
    }
    err = hipHostMalloc((void**)&E, vector_size);
    if (err != hipSuccess)
    {
        printf("Error allocating memory for E\n");
        hipHostFree(A);
        hipHostFree(B);
        hipHostFree(MC);
        hipHostFree(MD);
        return;
    }
    err = hipHostMalloc((void**)&Mtmp, matrix_size);
    if (err != hipSuccess)
    {
        printf("Error allocating memory for Mtmp\n");
        hipHostFree(A);
        hipHostFree(B);
        hipHostFree(MC);
        hipHostFree(MD);
        hipHostFree(E);
        return;
    }

    // Fill A, Mtmp
    fill_vector(A, SIZE_N);
    fill_matrix(Mtmp, SIZE_N);

    // Read data from files
    read_vector_float("data\\256_B_f.txt", B, SIZE_N);
    read_matrix_float("data\\256_MC_f.txt", MC, SIZE_N, SIZE_N);
    read_matrix_float("data\\256_MD_f.txt", MD, SIZE_N, SIZE_N);
    read_vector_float("data\\256_E_f.txt", E, SIZE_N);

    // Device pointers
    float* d_A = NULL;
    float* d_B = NULL;
    float* d_MC = NULL;
    float* d_MD = NULL;
    float* d_E = NULL;
    float* d_Mtmp = NULL;

    // Allocate memory on the device
    err = hipMalloc((void **)&d_A, vector_size);
    if (err != hipSuccess)
    {
        hipHostFree(A);
        hipHostFree(B);
        hipHostFree(MC);
        hipHostFree(MD);
        hipHostFree(E);
        hipHostFree(Mtmp);
        printf("Error allocating memory for A on device\n");
        return;
    }
    err = hipMalloc((void **)&d_B, vector_size);
    if (err != hipSuccess)
    {
        hipHostFree(A);
        hipHostFree(B);
        hipHostFree(MC);
        hipHostFree(MD);
        hipHostFree(E);
        hipHostFree(Mtmp);
        hipFree(d_A);
        printf("Error allocating memory for B on device\n");
        return;
    }
    err = hipMalloc((void **)&d_MC, matrix_size);
    if (err != hipSuccess)
    {
        hipHostFree(A);
        hipHostFree(B);
        hipHostFree(MC);
        hipHostFree(MD);
        hipHostFree(E);
        hipHostFree(Mtmp);
        hipFree(d_A);
        hipFree(d_B);
        printf("Error allocating memory for MC on device\n");
        return;
    }
    err = hipMalloc((void **)&d_MD, matrix_size);
    if (err != hipSuccess)
    {
        hipHostFree(A);
        hipHostFree(B);
        hipHostFree(MC);
        hipHostFree(MD);
        hipHostFree(E);
        hipHostFree(Mtmp);
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_MC);
        printf("Error allocating memory for MD on device\n");
        return;
    }
    err = hipMalloc((void **)&d_E, vector_size);
    if (err != hipSuccess)
    {
        hipHostFree(A);
        hipHostFree(B);
        hipHostFree(MC);
        hipHostFree(MD);
        hipHostFree(E);
        hipHostFree(Mtmp);
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_MC);
        hipFree(d_MD);
        printf("Error allocating memory for MD on device\n");
        return;
    }
    err = hipMalloc((void **)&d_Mtmp, matrix_size);
    if (err != hipSuccess)
    {
        hipHostFree(A);
        hipHostFree(B);
        hipHostFree(MC);
        hipHostFree(MD);
        hipHostFree(E);
        hipHostFree(Mtmp);
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_MC);
        hipFree(d_MD);
        hipFree(d_E);
        printf("Error allocating memory for Mtmp on device\n");
        return;
    }

    // Prepare for kernel launches
    int block_x = BLOCK_X;
    int block_y = BLOCK_Y;
    int block_z = BLOCK_Z;
    int grid_x = GRID_X;
    int grid_y = GRID_Y;
    int grid_z = GRID_Z;
    dim3 block_dims(block_x, block_y, block_z);
    dim3 grid_dims(grid_x, grid_y, grid_z);

    if (CUBLAS_ENABLE == 0)
    {
        printf("GPU - Using kernel calls.\n");
        // Copy data to the device
        err = hipMemcpyAsync(d_B, B, vector_size, hipMemcpyHostToDevice, stream);
        if (err != hipSuccess)
        {
            printf("Error copying B to the device.");
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_MC);
            hipFree(d_MD);
            hipFree(d_E);
            hipFree(d_Mtmp);
            hipHostFree(A);
            hipHostFree(B);
            hipHostFree(MC);
            hipHostFree(MD);
            hipHostFree(E);
            hipHostFree(Mtmp);
            return;
        }
        err = hipMemcpyAsync(d_MC, MC, matrix_size, hipMemcpyHostToDevice, stream);
        if (err != hipSuccess)
        {
            printf("Error copying MC to the device.");
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_MC);
            hipFree(d_MD);
            hipFree(d_E);
            hipFree(d_Mtmp);
            hipHostFree(A);
            hipHostFree(B);
            hipHostFree(MC);
            hipHostFree(MD);
            hipHostFree(E);
            hipHostFree(Mtmp);
            return;
        }
        err = hipMemcpyAsync(d_MD, MD, matrix_size, hipMemcpyHostToDevice, stream);
        if (err != hipSuccess)
        {
            printf("Error copying MD to the device.");
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_MC);
            hipFree(d_MD);
            hipFree(d_E);
            hipFree(d_Mtmp);
            hipHostFree(A);
            hipHostFree(B);
            hipHostFree(MC);
            hipHostFree(MD);
            hipHostFree(E);
            hipHostFree(Mtmp);
            return;
        }
        err = hipMemcpyAsync(d_E, E, vector_size, hipMemcpyHostToDevice, stream);
        if (err != hipSuccess)
        {
            printf("Error copying E to the device.");
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_MC);
            hipFree(d_MD);
            hipFree(d_E);
            hipFree(d_Mtmp);
            hipHostFree(A);
            hipHostFree(B);
            hipHostFree(MC);
            hipHostFree(MD);
            hipHostFree(E);
            hipHostFree(Mtmp);
            return;
        }
        err = hipMemcpyAsync(d_Mtmp, Mtmp, matrix_size, hipMemcpyHostToDevice, stream);
        if (err != hipSuccess)
        {
            printf("Error copying Mtmp to the device.");
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_MC);
            hipFree(d_MD);
            hipFree(d_E);
            hipFree(d_Mtmp);
            hipHostFree(A);
            hipHostFree(B);
            hipHostFree(MC);
            hipHostFree(MD);
            hipHostFree(E);
            hipHostFree(Mtmp);
            return;
        }

        // Launch kernels
        err = hipEventRecord(start, stream);
        if (err != hipSuccess)
    {
        printf("Error recording start event.");
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_MC);
        hipFree(d_MD);
        hipFree(d_E);
        hipFree(d_Mtmp);
        hipHostFree(A);
        hipHostFree(B);
        hipHostFree(MC);
        hipHostFree(MD);
        hipHostFree(E);
        hipHostFree(Mtmp);
        return;
    }
        f_gpu_matrix_multiply_matrix_acc<<<grid_dims, block_dims, 0, stream>>>(d_MC, d_MD, d_Mtmp, SIZE_N);
        f_gpu_matrix_multiply_vector_acc<<<grid_dims, block_dims, 0, stream>>>(d_Mtmp, d_E, d_A, SIZE_N);
        f_gpu_vector_add_vector<<<grid_dims, block_dims, 0, stream>>>(d_A, d_B, SIZE_N);

        err = hipStreamSynchronize(stream);
        if (err != hipSuccess)
    {
        printf("Error syncronizing stream after kernels.");
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_MC);
        hipFree(d_MD);
        hipFree(d_E);
        hipFree(d_Mtmp);
        hipHostFree(A);
        hipHostFree(B);
        hipHostFree(MC);
        hipHostFree(MD);
        hipHostFree(E);
        hipHostFree(Mtmp);
        return;
    }
        err = hipEventRecord(stop, stream);
        if (err != hipSuccess)
    {
        printf("Error recording stop event.");
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_MC);
        hipFree(d_MD);
        hipFree(d_E);
        hipFree(d_Mtmp);
        hipHostFree(A);
        hipHostFree(B);
        hipHostFree(MC);
        hipHostFree(MD);
        hipHostFree(E);
        hipHostFree(Mtmp);
        return;
    }
        err = hipEventSynchronize(stop);
        if (err != hipSuccess)
    {
        printf("Error recording stop event.");
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_MC);
        hipFree(d_MD);
        hipFree(d_E);
        hipFree(d_Mtmp);
        hipHostFree(A);
        hipHostFree(B);
        hipHostFree(MC);
        hipHostFree(MD);
        hipHostFree(E);
        hipHostFree(Mtmp);
        return;
    }

        // Copy result to the host
        err = hipMemcpy(A, d_A, vector_size, hipMemcpyDeviceToHost);
        if (err != hipSuccess)
        {
            printf("Error copying result to host.");
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_MC);
            hipFree(d_MD);
            hipFree(d_E);
            hipFree(d_Mtmp);
            hipHostFree(A);
            hipHostFree(B);
            hipHostFree(MC);
            hipHostFree(MD);
            hipHostFree(E);
            hipHostFree(Mtmp);
            return;
        }
    }

    if (CUBLAS_ENABLE == 1)
    {
        printf("GPU - Using cuBLAS calls.\n");
        // Variables for cuBLAS
        hipblasHandle_t h_cublas;
        hipblasStatus_t cublas_status;
        float alpha = 1.0f;
        float beta = 0.0f;

        // Init cuBLAS
        cublas_status = hipblasCreate(&h_cublas);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS)
        {
            printf("Error starting cuBLAS.");
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_MC);
            hipFree(d_MD);
            hipFree(d_E);
            hipFree(d_Mtmp);
            hipHostFree(A);
            hipHostFree(B);
            hipHostFree(MC);
            hipHostFree(MD);
            hipHostFree(E);
            hipHostFree(Mtmp);
            return;
        }

        // Set stream
        cublas_status = hipblasSetStream(h_cublas, stream);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS)
        {
            printf("Failed to set a stream.");
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_MC);
            hipFree(d_MD);
            hipFree(d_E);
            hipFree(d_Mtmp);
            hipHostFree(A);
            hipHostFree(B);
            hipHostFree(MC);
            hipHostFree(MD);
            hipHostFree(E);
            hipHostFree(Mtmp);
            return;
        }

        // Copy data to the device
        cublas_status = hipblasSetVectorAsync(SIZE_N, sizeof(float), B, 1, d_B, 1, stream);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS)
        {
            printf("Failed to copy vector B.");
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_MC);
            hipFree(d_MD);
            hipFree(d_E);
            hipFree(d_Mtmp);
            hipHostFree(A);
            hipHostFree(B);
            hipHostFree(MC);
            hipHostFree(MD);
            hipHostFree(E);
            hipHostFree(Mtmp);
            return;
        }
        cublas_status = hipblasSetMatrixAsync(SIZE_N, SIZE_N, sizeof(float), MC, SIZE_N, d_MC, SIZE_N, stream);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS)
        {
            printf("Failed to copy matrix MC.");
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_MC);
            hipFree(d_MD);
            hipFree(d_E);
            hipFree(d_Mtmp);
            hipHostFree(A);
            hipHostFree(B);
            hipHostFree(MC);
            hipHostFree(MD);
            hipHostFree(E);
            hipHostFree(Mtmp);
            return;
        }
        cublas_status = hipblasSetMatrixAsync(SIZE_N, SIZE_N, sizeof(float), MD, SIZE_N, d_MD, SIZE_N, stream);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS)
        {
            printf("Failed to copy matrix MD.");
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_MC);
            hipFree(d_MD);
            hipFree(d_E);
            hipFree(d_Mtmp);
            hipHostFree(A);
            hipHostFree(B);
            hipHostFree(MC);
            hipHostFree(MD);
            hipHostFree(E);
            hipHostFree(Mtmp);
            return;
        }
        cublas_status = hipblasSetVectorAsync(SIZE_N, sizeof(float), E, 1, d_E, 1, stream);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS)
        {
            printf("Failed to copy vector E.");
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_MC);
            hipFree(d_MD);
            hipFree(d_E);
            hipFree(d_Mtmp);
            hipHostFree(A);
            hipHostFree(B);
            hipHostFree(MC);
            hipHostFree(MD);
            hipHostFree(E);
            hipHostFree(Mtmp);
            return;
        }
        cublas_status = hipblasSetMatrixAsync(SIZE_N, SIZE_N, sizeof(float), Mtmp, SIZE_N, d_Mtmp, SIZE_N, stream);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS)
        {
            printf("Failed to copy matrix Mtmp.");
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_MC);
            hipFree(d_MD);
            hipFree(d_E);
            hipFree(d_Mtmp);
            hipHostFree(A);
            hipHostFree(B);
            hipHostFree(MC);
            hipHostFree(MD);
            hipHostFree(E);
            hipHostFree(Mtmp);
            return;
        }

        // Start the timer
        err = hipEventRecord(start, stream);
        if (err != hipSuccess)
        {
            printf("Error starting the timer.");
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_MC);
            hipFree(d_MD);
            hipFree(d_E);
            hipFree(d_Mtmp);
            hipHostFree(A);
            hipHostFree(B);
            hipHostFree(MC);
            hipHostFree(MD);
            hipHostFree(E);
            hipHostFree(Mtmp);
            return;
        }

        // Perform calculation
        cublas_status = hipblasSgemm( // MC = alpha (MA * MB) + beta * MC, S for single-precision
            h_cublas,
            HIPBLAS_OP_N, HIPBLAS_OP_N, // Don't transpose A and B
            SIZE_N, SIZE_N, SIZE_N, // Sizes of matrices
            &alpha,
            d_MC, SIZE_N, // MC
            d_MD, SIZE_N, // MD
            &beta,
            d_Mtmp, SIZE_N // Mtmp
        );
        if (cublas_status != HIPBLAS_STATUS_SUCCESS)
        {
            printf("Error performing MC * MD.");
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_MC);
            hipFree(d_MD);
            hipFree(d_E);
            hipFree(d_Mtmp);
            hipHostFree(A);
            hipHostFree(B);
            hipHostFree(MC);
            hipHostFree(MD);
            hipHostFree(E);
            hipHostFree(Mtmp);
            return;
        }

        cublas_status = hipblasSgemv( // A = alpha (Mtmp * E) + beta * A, S for single-precision
            h_cublas,
            HIPBLAS_OP_N,
            SIZE_N, SIZE_N,
            &alpha,
            d_Mtmp, SIZE_N, // MC * MD
            d_E, 1, // E
            &beta,
            d_A, 1 // A
        );

        if (cublas_status != HIPBLAS_STATUS_SUCCESS)
        {
            printf("Error performing Mtmp * E.");
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_MC);
            hipFree(d_MD);
            hipFree(d_E);
            hipFree(d_Mtmp);
            hipHostFree(A);
            hipHostFree(B);
            hipHostFree(MC);
            hipHostFree(MD);
            hipHostFree(E);
            hipHostFree(Mtmp);
            return;
        }

        f_gpu_vector_add_vector<<<grid_dims, block_dims, 0, stream>>>(d_A, d_B, SIZE_N);

        err = hipStreamSynchronize(stream);
        if (err != hipSuccess)
        {
            printf("Error syncronizing stream.");
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_MC);
            hipFree(d_MD);
            hipFree(d_E);
            hipFree(d_Mtmp);
            hipHostFree(A);
            hipHostFree(B);
            hipHostFree(MC);
            hipHostFree(MD);
            hipHostFree(E);
            hipHostFree(Mtmp);
            return;
        }
        err = hipEventRecord(stop, stream);
        if (err != hipSuccess)
        {
            printf("Error stopping the timer.");
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_MC);
            hipFree(d_MD);
            hipFree(d_E);
            hipFree(d_Mtmp);
            hipHostFree(A);
            hipHostFree(B);
            hipHostFree(MC);
            hipHostFree(MD);
            hipHostFree(E);
            hipHostFree(Mtmp);
            return;
        }
        err = hipEventSynchronize(stop);
        if (err != hipSuccess)
        {
            printf("Error syncronizing the timer.");
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_MC);
            hipFree(d_MD);
            hipFree(d_E);
            hipFree(d_Mtmp);
            hipHostFree(A);
            hipHostFree(B);
            hipHostFree(MC);
            hipHostFree(MD);
            hipHostFree(E);
            hipHostFree(Mtmp);
            return;
        }

        // Copy result to the host
        cublas_status = hipblasGetVector(SIZE_N, sizeof(float), d_A, 1, A, 1);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS)
        {
            printf("Failed to copy vector A.");
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_MC);
            hipFree(d_MD);
            hipFree(d_E);
            hipFree(d_Mtmp);
            hipHostFree(A);
            hipHostFree(B);
            hipHostFree(MC);
            hipHostFree(MD);
            hipHostFree(E);
            hipHostFree(Mtmp);
            return;
        }
        
        // Destroy cuBLAS
        cublas_status = hipblasDestroy(h_cublas);
        if (cublas_status != HIPBLAS_STATUS_SUCCESS)
        {
            printf("Error destroying cublas.");
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_MC);
            hipFree(d_MD);
            hipFree(d_E);
            hipFree(d_Mtmp);
            hipHostFree(A);
            hipHostFree(B);
            hipHostFree(MC);
            hipHostFree(MD);
            hipHostFree(E);
            hipHostFree(Mtmp);
            return;
        }
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_MC);
    hipFree(d_MD);
    hipFree(d_E);
    hipFree(d_Mtmp);

    // Calculate elapsed time
    float elapsed_time;
    err = hipEventElapsedTime(&elapsed_time, start, stop);
    if (err != hipSuccess)
    {
        printf("Error evaluating elapsed time.");
        return;
    }
    printf("Elapsed time: %.5f milliseconds\n", elapsed_time);
    
    // Print result
    print_vector_result(A, SIZE_N);
    // Write result to file
    write_vector_float("result\\result_gpu_prg2.txt", A, SIZE_N);

    // Free host memory
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(MC);
    hipHostFree(MD);
    hipHostFree(E);
    hipHostFree(Mtmp);

    return;
}
