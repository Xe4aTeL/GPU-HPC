#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "gpu_kernel.cuh"
#include "cpu_math.h"
#include "file_handler.h"
#include "config.h"


void fill_matrix(int* matrix, int N);
void fill_vector(int* vector, int N);
void print_matrix_result(int* matrix, int N);
void print_vector_result(int* vector, int N);
void cpu_mode();
void gpu_mode();

int main(int argc, char* argv[])
{
    // Check if the program is run with the correct number of arguments
    if (argc != 2)
    {
        printf("Usage: %s <GPU_ENABLE>\n", argv[0]);
        return -1;
    }

    // Parse GPU_ENABLE from command line argument
    int GPU_ENABLE = atoi(argv[1]);

    // Check if GPU_ENABLE is valid
    if (GPU_ENABLE != 0 && GPU_ENABLE != 1)
    {
        printf("Error: GPU_ENABLE must be 0 or 1\n");
        return -1;
    }

    // CPU mode memory allocation
    if (GPU_ENABLE == 0)
    {
        cpu_mode();
    }

    // GPU mode memory allocation
    if (GPU_ENABLE == 1)
    {
        gpu_mode();
    }

    printf("Done\n");
    return 0;
}

void fill_matrix(int* matrix, int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            matrix[i * N + j] = 0;
        }
    }
}

void fill_vector(int* vector, int N)
{
    for (int i = 0; i < N; i++)
    {
        vector[i] = 0;
    }
}

void print_matrix_result(int* matrix, int N)
{
    if (N > 32)
    {
        printf("Result is too large to print.\n");
        return;
    }

    printf("Result: ");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%d ", matrix[i * N + j]);
        }
        printf("\n");
    }

    printf("\n");
}

void print_vector_result(int* vector, int N)
{
    if (N > 32)
    {
        printf("Result is too large to print.\n");
        return;
    }

    printf("Result: ");
    for (int i = 0; i < N; i++)
    {
        printf("%d ", vector[i]);
    }

    printf("\n");
}

void cpu_mode()
{
    printf("CPU mode\n");
    // Variables
    int matrix_size = SIZE_N * SIZE_N * sizeof(int);
    int *MA, *MB, *MC, *ME;
    int d;

    // Malloc
    MA = (int*)malloc(matrix_size);
    if (MA == NULL)
    {
        printf("Error allocating memory for MA\n");
        return;
    }
    MB = (int*)malloc(matrix_size);
    if (MB == NULL)
    {
        printf("Error allocating memory for MB\n");
        free(MA);
        return;
    }
    MC = (int*)malloc(matrix_size);
    if (MC == NULL)
    {
        printf("Error allocating memory for MC\n");
        free(MA);
        free(MB);
        return;
    }
    ME = (int*)malloc(matrix_size);
    if (ME == NULL)
    {
        printf("Error allocating memory for ME\n");
        free(MA);
        free(MB);
        free(MC);
        return;
    }

    // Fill MA
    fill_matrix(MA, SIZE_N);

    // Read data from files
    read_matrix_int("data\\8192_MB_i.txt", MB, SIZE_N, SIZE_N);
    read_matrix_int("data\\8192_MC_i.txt", MC, SIZE_N, SIZE_N);
    read_matrix_int("data\\8192_ME_i.txt", ME, SIZE_N, SIZE_N);
    read_scalar_int("data\\d_i.txt", &d);

    // Timer + Start
    struct timespec start, end;
    timespec_get(&start, TIME_UTC);

    // MA = MB * MC + d * ME
    i_matrix_multiply_matrix_acc(MB, MC, MA, SIZE_N);
    i_matrix_multiply_scalar_acc(ME, d, MA, SIZE_N);

    // End timer
    timespec_get(&end, TIME_UTC);

    // Show elapsed time
    double elapsed_time = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1e9;
    printf("Elapsed time: %.5f milliseconds\n", elapsed_time * 1000);

    // Print and save result
    print_matrix_result(MA, SIZE_N);
    write_matrix_int("result\\result_cpu_prg1.txt", MA, SIZE_N, SIZE_N);

    // Free malloc
    free(MA);
    free(MB);
    free(MC);
    free(ME);
    MA = NULL;
    MB = NULL;
    MC = NULL;
    ME = NULL;

    return;
}

void gpu_mode()
{
    if (THREADS_PER_BLOCK > 1024)
    {
        printf("Error: THREADS_PER_BLOCK exceeds 1024\n");
        return;
    }
    if (GRID_Y > 65535 || BLOCKS_PER_GRID > 2147483647) // "2 ^ 31 - 1" or "(1 << 31) - 1"
    {
        printf("Error: BLOCK_Y exceeds 65535 or BLOCKS_PER_GRID exceeds 2 ^ 31 - 1\n");
        return;
    }

    printf("GPU mode\n");
    // Variables
    int matrix_size = SIZE_N * SIZE_N * sizeof(int);
    int *MA, *MB, *MC, *ME;
    int d;
    hipError_t err;

    // CUDA events for performance measurement
    hipEvent_t start, stop;
    err = hipEventCreate(&start);
    if (err != hipSuccess)
    {
        printf("Error creating event for timing.");
        return;
    }
    err = hipEventCreate(&stop);
    if (err != hipSuccess)
    {
        printf("Error creating event for timing.");
        return;
    }

    hipStream_t stream;
    err = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    if (err != hipSuccess)
    {
        printf("Error creating stream.");
        return;
    }

    // Allocate memory (Host)
    err = hipHostMalloc((void**)&MA, matrix_size);
    if (err != hipSuccess)
    {
        printf("Error allocating memory for MA\n");
        return;
    }
    err = hipHostMalloc((void**)&MB, matrix_size);
    if (err != hipSuccess)
    {
        printf("Error allocating memory for MB\n");
        hipHostFree(MA);
        return;
    }
    err = hipHostMalloc((void**)&MC, matrix_size);
    if (err != hipSuccess)
    {
        printf("Error allocating memory for MC\n");
        hipHostFree(MA);
        hipHostFree(MB);
        return;
    }
    err = hipHostMalloc((void**)&ME, matrix_size);
    if (err != hipSuccess)
    {
        printf("Error allocating memory for ME\n");
        hipHostFree(MA);
        hipHostFree(MB);
        hipHostFree(MC);
        return;
    }

    // Fill MA
    fill_matrix(MA, SIZE_N);

    // Read data from files
    read_matrix_int("data\\8192_MB_i.txt", MB, SIZE_N, SIZE_N);
    read_matrix_int("data\\8192_MC_i.txt", MC, SIZE_N, SIZE_N);
    read_matrix_int("data\\8192_ME_i.txt", ME, SIZE_N, SIZE_N);
    read_scalar_int("data\\d_i.txt", &d);

    // Device pointers
    int* d_MA = NULL;
    int* d_MB = NULL;
    int* d_MC = NULL;
    int* d_ME = NULL;
    int* d_d = NULL;

    // Allocate memory on the device
    err = hipMalloc((void **)&d_MA, matrix_size);
    if (err != hipSuccess)
    {
        hipHostFree(MA);
        hipHostFree(MB);
        hipHostFree(MC);
        hipHostFree(ME);
        printf("Error allocating memory for MA on device\n");
        return;
    }
    err = hipMalloc((void **)&d_MB, matrix_size);
    if (err != hipSuccess)
    {
        hipHostFree(MA);
        hipHostFree(MB);
        hipHostFree(MC);
        hipHostFree(ME);
        hipFree(d_MA);
        printf("Error allocating memory for MB on device\n");
        return;
    }
    err = hipMalloc((void **)&d_MC, matrix_size);
    if (err != hipSuccess)
    {
        hipHostFree(MA);
        hipHostFree(MB);
        hipHostFree(MC);
        hipHostFree(ME);
        hipFree(d_MA);
        hipFree(d_MB);
        printf("Error allocating memory for MC on device\n");
        return;
    }
    err = hipMalloc((void **)&d_ME, matrix_size);
    if (err != hipSuccess)
    {
        hipHostFree(MA);
        hipHostFree(MB);
        hipHostFree(MC);
        hipHostFree(ME);
        hipFree(d_MA);
        hipFree(d_MB);
        hipFree(d_MC);
        printf("Error allocating memory for ME on device\n");
        return;
    }
    err = hipMalloc((void **)&d_d, sizeof(int));
    if (err != hipSuccess)
    {
        hipHostFree(MA);
        hipHostFree(MB);
        hipHostFree(MC);
        hipHostFree(ME);
        hipFree(d_MA);
        hipFree(d_MB);
        hipFree(d_MC);
        hipFree(d_ME);
        printf("Error allocating memory for d on device\n");
        return;
    }


    // Copy data to the device
    err = hipMemcpyAsync(d_MB, MB, matrix_size, hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        printf("Error copying MB to the device.");
        hipFree(d_MA);
        hipFree(d_MB);
        hipFree(d_MC);
        hipFree(d_d);
        hipFree(d_ME);
        hipHostFree(MA);
        hipHostFree(MB);
        hipHostFree(MC);
        hipHostFree(ME);
        return;
    }
    err = hipMemcpyAsync(d_MC, MC, matrix_size, hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        printf("Error copying MC to the device.");
        hipFree(d_MA);
        hipFree(d_MB);
        hipFree(d_MC);
        hipFree(d_d);
        hipFree(d_ME);
        hipHostFree(MA);
        hipHostFree(MB);
        hipHostFree(MC);
        hipHostFree(ME);
        return;
    }
    err = hipMemcpyAsync(d_ME, ME, matrix_size, hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        printf("Error copying ME to the device.");
        hipFree(d_MA);
        hipFree(d_MB);
        hipFree(d_MC);
        hipFree(d_d);
        hipFree(d_ME);
        hipHostFree(MA);
        hipHostFree(MB);
        hipHostFree(MC);
        hipHostFree(ME);
        return;
    }
    err = hipMemcpyAsync(d_d, &d, sizeof(d), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess)
    {
        printf("Error copying d to the device.");
        hipFree(d_MA);
        hipFree(d_MB);
        hipFree(d_MC);
        hipFree(d_d);
        hipFree(d_ME);
        hipHostFree(MA);
        hipHostFree(MB);
        hipHostFree(MC);
        hipHostFree(ME);
        return;
    }
    
    // Prepare for kernel launches
    int block_x = BLOCK_X;
    int block_y = BLOCK_Y;
    int block_z = BLOCK_Z;
    int grid_x = GRID_X;
    int grid_y = GRID_Y;
    int grid_z = GRID_Z;
    dim3 block_dims(block_x, block_y, block_z);
    dim3 grid_dims(grid_x, grid_y, grid_z);

    // Launch kernels
    err = hipStreamSynchronize(stream);
    if (err != hipSuccess)
    {
        printf("Error syncronizing stream before kernels.");
        hipFree(d_MA);
        hipFree(d_MB);
        hipFree(d_MC);
        hipFree(d_d);
        hipFree(d_ME);
        hipHostFree(MA);
        hipHostFree(MB);
        hipHostFree(MC);
        hipHostFree(ME);
        return;
    }
    err = hipEventRecord(start, stream);
    if (err != hipSuccess)
    {
        printf("Error recording start event.");
        hipFree(d_MA);
        hipFree(d_MB);
        hipFree(d_MC);
        hipFree(d_d);
        hipFree(d_ME);
        hipHostFree(MA);
        hipHostFree(MB);
        hipHostFree(MC);
        hipHostFree(ME);
        return;
    }
    i_gpu_matrix_multiply_matrix_acc<<<grid_dims, block_dims, 0, stream>>>(d_MB, d_MC, d_MA, SIZE_N);
    i_gpu_matrix_multiply_scalar_acc<<<grid_dims, block_dims, 0, stream>>>(d_ME, d_d, d_MA, SIZE_N);

    err = hipStreamSynchronize(stream);
    if (err != hipSuccess)
    {
        printf("Error syncronizing stream after kernels.");
        hipFree(d_MA);
        hipFree(d_MB);
        hipFree(d_MC);
        hipFree(d_d);
        hipFree(d_ME);
        hipHostFree(MA);
        hipHostFree(MB);
        hipHostFree(MC);
        hipHostFree(ME);
        return;
    }
    err = hipEventRecord(stop, stream);
    if (err != hipSuccess)
    {
        printf("Error recording stop event.");
        hipFree(d_MA);
        hipFree(d_MB);
        hipFree(d_MC);
        hipFree(d_d);
        hipFree(d_ME);
        hipHostFree(MA);
        hipHostFree(MB);
        hipHostFree(MC);
        hipHostFree(ME);
        return;
    }

    // Copy result to the host
    err = hipMemcpy(MA, d_MA, matrix_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Error coyping result to host.");
        hipFree(d_MA);
        hipFree(d_MB);
        hipFree(d_MC);
        hipFree(d_d);
        hipFree(d_ME);
        hipHostFree(MA);
        hipHostFree(MB);
        hipHostFree(MC);
        hipHostFree(ME);
        return;
    }

    hipEventSynchronize(stop);

    // Free device memory
    hipFree(d_MA);
    hipFree(d_MB);
    hipFree(d_MC);
    hipFree(d_d);
    hipFree(d_ME);

    // Calculate elapsed time
    float elapsed_time;
    err = hipEventElapsedTime(&elapsed_time, start, stop);
    if (err != hipSuccess)
    {
        printf("Error evaluating elapsed time.");
        return;
    }
    printf("Elapsed time: %.5f milliseconds\n", elapsed_time);
    
    // Print result
    print_matrix_result(MA, SIZE_N);
    // Write result to file
    write_matrix_int("result\\result_gpu_prg1.txt", MA, SIZE_N, SIZE_N);

    // Free host memory
    hipHostFree(MA);
    hipHostFree(MB);
    hipHostFree(MC);
    hipHostFree(ME);

    return;
}
