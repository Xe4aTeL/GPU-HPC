/*
PRG1 - Програма 1
Задача:
MA = MB * MC + d * ME

Виконав:
Скоробагатько Іван ІО-13

PRG1 - Program 1
Task:
MA = MB * MC + d * ME

Programmed by:
Skorobagatko Ivan ІО-13
*/

#include <time.h>

#include "file_handler.h"
#include "gpu_kernel.cuh"
#include "cpu_math.h"
#include "config.h"


void fill_matrix(int* matrix, int N);
void print_matrix_result(int* matrix, int N);
void cpu_mode(int SIZE_N);
void gpu_mode(int SIZE_N);

int main(int argc, char* argv[]) {
    // Check if the program is run with the correct number of arguments
    if (argc != 3) {
        printf("Usage: <GPU_ENABLE> <SIZE>\n");
        exit(EXIT_FAILURE);
    }

    // Parse GPU_ENABLE from command line argument
    int GPU_ENABLE = atoi(argv[1]);

    // Parse SIZE_N from command line argument
    int SIZE_N = atoi(argv[2]);

    // Check if GPU_ENABLE is valid
    if (GPU_ENABLE != 0 && GPU_ENABLE != 1) {
        printf("Error: GPU_ENABLE must be 0 or 1\n");
        exit(EXIT_FAILURE);
    }

    // CPU mode memory allocation
    if (GPU_ENABLE == 0)
        cpu_mode(SIZE_N);

    // GPU mode memory allocation
    if (GPU_ENABLE == 1)
        gpu_mode(SIZE_N);

    printf("Done\n");
    exit(EXIT_SUCCESS);
}

void fill_matrix(int* matrix, int N) {
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
            matrix[i * N + j] = 0;
}

void print_matrix_result(int* matrix, int N) {
    if (N > 32) {
        printf("Result is too large to print.\n");
        return;
    }

    printf("Result: ");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++)
            printf("%d ", matrix[i * N + j]);
        printf("\n");
    }
    printf("\n");
}

void cpu_mode(int SIZE_N) {
    printf("CPU mode\n");
    // Variable, timespec for time measurement
    int *MA, *MB, *MC, *ME;
    const int matrix_size = SIZE_N * SIZE_N * sizeof(int);
    int d;
    struct timespec start, end;

    // Malloc
    MA = (int*)malloc(matrix_size);
    if (MA == NULL) {
        printf("Error allocating memory for MA\n");
        exit(EXIT_FAILURE);
    }
    MB = (int*)malloc(matrix_size);
    if (MB == NULL) {
        printf("Error allocating memory for MB\n");
        free(MA);
        exit(EXIT_FAILURE);
    }
    MC = (int*)malloc(matrix_size);
    if (MC == NULL) {
        printf("Error allocating memory for MC\n");
        free(MA);
        free(MB);
        exit(EXIT_FAILURE);
    }
    ME = (int*)malloc(matrix_size);
    if (ME == NULL) {
        printf("Error allocating memory for ME\n");
        free(MA);
        free(MB);
        free(MC);
        exit(EXIT_FAILURE);
    }

    // Fill MA
    fill_matrix(MA, SIZE_N);

    // Read data from files
    switch (SIZE_N) {
        case 256:
            read_matrix_int("data\\256_MB_i.txt", MB, SIZE_N, SIZE_N);
            read_matrix_int("data\\256_MC_i.txt", MC, SIZE_N, SIZE_N);
            read_matrix_int("data\\256_ME_i.txt", ME, SIZE_N, SIZE_N);
            read_scalar_int("data\\d_i.txt", &d);
            break;
        case 512:
            read_matrix_int("data\\512_MB_i.txt", MB, SIZE_N, SIZE_N);
            read_matrix_int("data\\512_MC_i.txt", MC, SIZE_N, SIZE_N);
            read_matrix_int("data\\512_ME_i.txt", ME, SIZE_N, SIZE_N);
            read_scalar_int("data\\d_i.txt", &d);
            break;
        case 1024:
            read_matrix_int("data\\1024_MB_i.txt", MB, SIZE_N, SIZE_N);
            read_matrix_int("data\\1024_MC_i.txt", MC, SIZE_N, SIZE_N);
            read_matrix_int("data\\1024_ME_i.txt", ME, SIZE_N, SIZE_N);
            read_scalar_int("data\\d_i.txt", &d);
            break;
        case 2048:
            read_matrix_int("data\\2048_MB_i.txt", MB, SIZE_N, SIZE_N);
            read_matrix_int("data\\2048_MC_i.txt", MC, SIZE_N, SIZE_N);
            read_matrix_int("data\\2048_ME_i.txt", ME, SIZE_N, SIZE_N);
            read_scalar_int("data\\d_i.txt", &d);
            break;
        case 4096:
            read_matrix_int("data\\4096_MB_i.txt", MB, SIZE_N, SIZE_N);
            read_matrix_int("data\\4096_MC_i.txt", MC, SIZE_N, SIZE_N);
            read_matrix_int("data\\4096_ME_i.txt", ME, SIZE_N, SIZE_N);
            read_scalar_int("data\\d_i.txt", &d);
            break;
        case 8192:
            read_matrix_int("data\\8192_MB_i.txt", MB, SIZE_N, SIZE_N);
            read_matrix_int("data\\8192_MC_i.txt", MC, SIZE_N, SIZE_N);
            read_matrix_int("data\\8192_ME_i.txt", ME, SIZE_N, SIZE_N);
            read_scalar_int("data\\d_i.txt", &d);
            break;
        case 16384:
            read_matrix_int("data\\16384_MB_i.txt", MB, SIZE_N, SIZE_N);
            read_matrix_int("data\\16384_MC_i.txt", MC, SIZE_N, SIZE_N);
            read_matrix_int("data\\16384_ME_i.txt", ME, SIZE_N, SIZE_N);
            read_scalar_int("data\\d_i.txt", &d);
            break;
        default:
            printf("Unusual size of input data, reading MB_i.txt, MC_i.txt, ME_i.txt");
            read_matrix_int("data\\MB_i.txt", MB, SIZE_N, SIZE_N);
            read_matrix_int("data\\MC_i.txt", MC, SIZE_N, SIZE_N);
            read_matrix_int("data\\ME_i.txt", ME, SIZE_N, SIZE_N);
            read_scalar_int("data\\d_i.txt", &d);
            break;
    }

    // Start timer
    timespec_get(&start, TIME_UTC);

    // MA = MB * MC + d * ME
    i_matrix_multiply_matrix_acc(MB, MC, MA, SIZE_N);
    i_matrix_multiply_scalar_acc(ME, d, MA, SIZE_N);

    // End timer
    timespec_get(&end, TIME_UTC);

    // Show elapsed time
    double elapsed_time = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1e9;
    printf("Elapsed time: %.5f milliseconds\n", elapsed_time * 1000);

    // Print and save result
    print_matrix_result(MA, SIZE_N);
    write_matrix_int("result\\result_cpu_prg1.txt", MA, SIZE_N, SIZE_N);

    // Free malloc
    free(MA);
    free(MB);
    free(MC);
    free(ME);
}

void gpu_mode(int SIZE_N) {
    // Configuration checks
    if (THREADS_PER_BLOCK > 1024) {
        printf("Error: THREADS_PER_BLOCK exceeds 1024\n");
        return;
    }
    if (GRID_Y > 65535 || BLOCKS_PER_GRID > 2147483647) // "2 ^ 31 - 1" or "(1 << 31) - 1"
    {
        printf("Error: BLOCK_Y exceeds 65535 or BLOCKS_PER_GRID exceeds 2 ^ 31 - 1\n");
        return;
    }

    printf("GPU mode\n");
    // Variables
    int *MA, *MB, *MC, *ME;
    const int matrix_size = SIZE_N * SIZE_N * sizeof(int);
    int d;
    hipError_t err;

    // CUDA events for performance measurement
    hipEvent_t start, stop;
    err = hipEventCreate(&start);
    if (err != hipSuccess) {
        printf("Error creating event for timing.");
        exit(EXIT_FAILURE);
    }
    err = hipEventCreate(&stop);
    if (err != hipSuccess) {
        printf("Error creating event for timing.");
        exit(EXIT_FAILURE);
    }

    // Allocate memory (Host)
    err = hipHostMalloc((void**)&MA, matrix_size);
    if (err != hipSuccess) {
        printf("Error allocating memory for MA\n");
        exit(EXIT_FAILURE);
    }
    err = hipHostMalloc((void**)&MB, matrix_size);
    if (err != hipSuccess) {
        printf("Error allocating memory for MB\n");
        exit(EXIT_FAILURE);
    }
    err = hipHostMalloc((void**)&MC, matrix_size);
    if (err != hipSuccess) {
        printf("Error allocating memory for MC\n");
        exit(EXIT_FAILURE);
    }
    err = hipHostMalloc((void**)&ME, matrix_size);
    if (err != hipSuccess) {
        printf("Error allocating memory for ME\n");
        exit(EXIT_FAILURE);
    }

    // Fill MA
    fill_matrix(MA, SIZE_N);

    // Read data from files
    switch (SIZE_N) {
        case 256:
            read_matrix_int("data\\256_MB_i.txt", MB, SIZE_N, SIZE_N);
            read_matrix_int("data\\256_MC_i.txt", MC, SIZE_N, SIZE_N);
            read_matrix_int("data\\256_ME_i.txt", ME, SIZE_N, SIZE_N);
            read_scalar_int("data\\d_i.txt", &d);
            break;
        case 512:
            read_matrix_int("data\\512_MB_i.txt", MB, SIZE_N, SIZE_N);
            read_matrix_int("data\\512_MC_i.txt", MC, SIZE_N, SIZE_N);
            read_matrix_int("data\\512_ME_i.txt", ME, SIZE_N, SIZE_N);
            read_scalar_int("data\\d_i.txt", &d);
            break;
        case 1024:
            read_matrix_int("data\\1024_MB_i.txt", MB, SIZE_N, SIZE_N);
            read_matrix_int("data\\1024_MC_i.txt", MC, SIZE_N, SIZE_N);
            read_matrix_int("data\\1024_ME_i.txt", ME, SIZE_N, SIZE_N);
            read_scalar_int("data\\d_i.txt", &d);
            break;
        case 2048:
            read_matrix_int("data\\2048_MB_i.txt", MB, SIZE_N, SIZE_N);
            read_matrix_int("data\\2048_MC_i.txt", MC, SIZE_N, SIZE_N);
            read_matrix_int("data\\2048_ME_i.txt", ME, SIZE_N, SIZE_N);
            read_scalar_int("data\\d_i.txt", &d);
            break;
        case 4096:
            read_matrix_int("data\\4096_MB_i.txt", MB, SIZE_N, SIZE_N);
            read_matrix_int("data\\4096_MC_i.txt", MC, SIZE_N, SIZE_N);
            read_matrix_int("data\\4096_ME_i.txt", ME, SIZE_N, SIZE_N);
            read_scalar_int("data\\d_i.txt", &d);
            break;
        case 8192:
            read_matrix_int("data\\8192_MB_i.txt", MB, SIZE_N, SIZE_N);
            read_matrix_int("data\\8192_MC_i.txt", MC, SIZE_N, SIZE_N);
            read_matrix_int("data\\8192_ME_i.txt", ME, SIZE_N, SIZE_N);
            read_scalar_int("data\\d_i.txt", &d);
            break;
        case 16384:
            read_matrix_int("data\\16384_MB_i.txt", MB, SIZE_N, SIZE_N);
            read_matrix_int("data\\16384_MC_i.txt", MC, SIZE_N, SIZE_N);
            read_matrix_int("data\\16384_ME_i.txt", ME, SIZE_N, SIZE_N);
            read_scalar_int("data\\d_i.txt", &d);
            break;
        default:
            printf("Unusual size of input data, reading MB_i.txt, MC_i.txt, ME_i.txt");
            read_matrix_int("data\\MB_i.txt", MB, SIZE_N, SIZE_N);
            read_matrix_int("data\\MC_i.txt", MC, SIZE_N, SIZE_N);
            read_matrix_int("data\\ME_i.txt", ME, SIZE_N, SIZE_N);
            read_scalar_int("data\\d_i.txt", &d);
            break;
    }

    // Device pointers
    int* d_MA = NULL;
    int* d_MB = NULL;
    int* d_MC = NULL;
    int* d_ME = NULL;
    int* d_d = NULL;

    // Allocate memory on the device
    err = hipMalloc((void **)&d_MA, matrix_size);
    if (err != hipSuccess) {
        printf("Error allocating memory for MA on device\n");
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&d_MB, matrix_size);
    if (err != hipSuccess) {
        printf("Error allocating memory for MB on device\n");
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&d_MC, matrix_size);
    if (err != hipSuccess) {
        printf("Error allocating memory for MC on device\n");
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&d_ME, matrix_size);
    if (err != hipSuccess) {
        printf("Error allocating memory for ME on device\n");
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&d_d, sizeof(int));
    if (err != hipSuccess) {
        printf("Error allocating memory for d on device\n");
        exit(EXIT_FAILURE);
    }

    // Copy data to the device
    err = hipMemcpyAsync(d_MB, MB, matrix_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Error copying MB to the device.");
        exit(EXIT_FAILURE);
    }
    err = hipMemcpyAsync(d_MC, MC, matrix_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Error copying MC to the device.");
        exit(EXIT_FAILURE);
    }
    err = hipMemcpyAsync(d_ME, ME, matrix_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Error copying ME to the device.");
        exit(EXIT_FAILURE);
    }
    err = hipMemcpyAsync(d_d, &d, sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Error copying d to the device.");
        exit(EXIT_FAILURE);
    }
    
    // Prepare for kernel launches
    dim3 block_dims(BLOCK_X, BLOCK_Y, BLOCK_Z);
    dim3 grid_dims(GRID_X, GRID_Y, GRID_Z);

    // Launch kernels
    err = hipEventRecord(start);
    if (err != hipSuccess) {
        printf("Error recording start event.");
        exit(EXIT_FAILURE);
    }

    i_gpu_matrix_multiply_matrix_acc<<<grid_dims, block_dims>>>(d_MB, d_MC, d_MA, SIZE_N);
    i_gpu_matrix_multiply_scalar_acc<<<grid_dims, block_dims>>>(d_ME, d_d, d_MA, SIZE_N);

    err = hipEventRecord(stop);
    if (err != hipSuccess) {
        printf("Error recording stop event.");
        exit(EXIT_FAILURE);
    }
    hipEventSynchronize(stop);
    if (err != hipSuccess) {
        printf("Error syncronizing.");
        exit(EXIT_FAILURE);
    }

    // Copy result to the host
    err = hipMemcpyAsync(MA, d_MA, matrix_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("Error copying result to host.");
        exit(EXIT_FAILURE);
    }

    // Free device memory
    hipFree(d_MA);
    hipFree(d_MB);
    hipFree(d_MC);
    hipFree(d_ME);

    // Show elapsed time
    float elapsed_time;
    err = hipEventElapsedTime(&elapsed_time, start, stop);
    if (err != hipSuccess) {
        printf("Error evaluating elapsed time.");
        exit(EXIT_FAILURE);
    }
    printf("Elapsed time: %.5f milliseconds\n", elapsed_time);
    
    // Print and save result
    print_matrix_result(MA, SIZE_N);
    write_matrix_int("result\\result_gpu_prg1.txt", MA, SIZE_N, SIZE_N);

    // Free host memory
    hipHostFree(MA);
    hipHostFree(MB);
    hipHostFree(MC);
    hipHostFree(ME);
}
